#include "hip/hip_runtime.h"
#include "harmonize.git/harmonize/cpp/harmonize.h"
#include <fstream>
#include <iostream>
#include <map>
#include <sstream>
#include <stdio.h>
#include <string>
#include <vector>
using namespace util;

typedef struct {
  int id;
  size_t edge_count;
  size_t edge_offset;
  unsigned int depth; // compare depth of this node and incoming depth
} Node;

// state that will be stored per program instance and accessible by all work
// groups immutable, but can contain references and pointers to non-const data
struct MyDeviceState {
  Node* node_arr;
  int node_count;
  int* edge_arr;
  int root_node;
  bool verbose;
  iter::AtomicIter<unsigned int>* iterator;
};

struct MyProgramOp {
  using Type = void (*)(Node* node, unsigned int current_depth, Node* parent);

  template <typename PROGRAM>
  __device__ static void eval(PROGRAM prog, Node *node, unsigned int current_depth, Node* parent) {
    int this_id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int orig_depth = node->depth;

    // if this node is already visited, then skip it
    if (atomicMin(&node->depth, current_depth) <= current_depth) {
      return; // base case
    }

    if (prog.device.verbose) {
      if (parent == nullptr) printf("[%d] node root->%d, depth %u->%u\n", this_id, node->id, orig_depth, node->depth);
      else printf("[%d] node %d->%d, depth %u->%u\n", this_id, parent->id, node->id, orig_depth, node->depth);
    }

    for (int i = 0; i < node->edge_count; i++) {
      int edge_node_id = prog.device.edge_arr[node->edge_offset + i];
      Node& edge_node = prog.device.node_arr[edge_node_id];
      prog.template async<MyProgramOp>(&edge_node, current_depth + 1, node);
    }
  }
};

struct MyProgramSpec {
  typedef OpUnion<MyProgramOp> OpSet;
  typedef MyDeviceState DeviceState;

  static const size_t STASH_SIZE =   16;
  static const size_t FRAME_SIZE = 8191;
  static const size_t POOL_SIZE  = 8191;

  /*
    type PROGRAM {
      device: DeviceState
      template: Op
    }
  */

  // called by each work group at start
  template <typename PROGRAM> __device__ static void initialize(PROGRAM prog) {}

  // called by each work group at end
  template <typename PROGRAM> __device__ static void finalize(PROGRAM prog) {}

  // called by each work group if need work
  template <typename PROGRAM> __device__ static bool make_work(PROGRAM prog) {
    unsigned int index;
    if (prog.device.iterator->step(index)) {
      prog.template async<MyProgramOp>(&prog.device.node_arr[prog.device.root_node], 0, nullptr);
    }

    return false;
  }
};

using ProgType = AsyncProgram<MyProgramSpec>;

int main(int argc, char *argv[]) {
  cli::ArgSet args(argc, argv);

  // arguments
  unsigned int batch_count = args["batch_count"] | 1;
  unsigned int run_count = args["run_count"] | 1;
  unsigned int arena_size = args["arena_size"] | 0x10000;
  std::string file_str = args.get_flag_str((char *)"file");

  // if flag is present, then true, else false
  bool directed = args["directed"];

  // init DeviceState
  MyDeviceState ds;
  ds.node_count = 0;
  ds.root_node = args["root"]; // int
  ds.verbose = args["verbose"]; // bool

  if (ds.verbose) {
    std::cout << "group count: " << batch_count << std::endl;
    std::cout << "cycle count: " << run_count << std::endl;
    std::cout << "arena size: " << arena_size << std::endl;
    std::cout << "parsing " << file_str << std::endl;
  }

  iter::AtomicIter<unsigned int> host_iter(0, 1);
  host::DevBuf<iter::AtomicIter<unsigned int>> iterator;
  iterator << host_iter;
  ds.iterator = iterator;

  std::vector<Node> nodes;
  std::map<int, std::vector<int>> adjacency_graph;

  std::ifstream file(file_str);
  if (!file.is_open()) {
    std::cerr << "unable to open " << file_str << std::endl;
    return 1;
  }

  std::string line;
  unsigned int line_idx = 0;
  while (std::getline(file, line)) {
    line_idx++;
    if (line.substr(0, 2) == "%%") {
      line_idx--; // trigger line_idx == 1
    } else if (line_idx == 1) {
      std::string token;
      std::stringstream ss(line);

      // parse node count
      getline(ss, token, ' ');
      ds.node_count = std::stoi(token) + 1;
      if (ds.verbose) {
        std::cout << "loading " << token << " nodes" << std::endl;
      }

      nodes = std::vector<Node>(
        ds.node_count, // 
        {.edge_count = 0, .edge_offset = 0, .depth = 0xFFFFFFFF}
      );

      for (size_t i = 0; i < nodes.size(); i++) {
        nodes.at(i).id = i;
      }
    } else {
      int node_id, edge;
      std::string token;
      std::stringstream ss(line);

      // parse node
      ss >> node_id;

      // parse edge
      ss >> edge;
      adjacency_graph[node_id].push_back(edge);

      if (!directed) {
        adjacency_graph[edge].push_back(node_id);
      }
    }
  }

  // finally close file
  file.close();

  // single edge array
  std::vector<int> edges;

  for (std::map<int, std::vector<int>>::iterator it = adjacency_graph.begin(); it != adjacency_graph.end(); it++) {
    size_t offset = edges.size(); // size before adding edges

    for (auto &&edge : it->second) {
      edges.push_back(edge);
    }

    Node node = {.id = it->first,
                 .edge_count = it->second.size(),
                 .edge_offset = offset,
                 .depth = 0xFFFFFFFF};
    nodes.at(node.id) = node;
  }

  host::DevBuf<int> dev_edges(edges.size());
  dev_edges << edges;
  ds.edge_arr = dev_edges;

  host::DevBuf<Node> dev_nodes(ds.node_count);
  dev_nodes << nodes;
  ds.node_arr = dev_nodes;

  if (ds.node_count == 0) {
    std::cerr << "error: node count = 0" << std::endl;
    return 0;
  }

  // declare program instance
  ProgType::Instance instance(arena_size, ds);
  hipDeviceSynchronize();
  host::check_error();

  // init program instance
  init<ProgType>(instance, 32);
  hipDeviceSynchronize();
  host::check_error();

  // exec program instance
  do {
    // Give the number of work groups and the size of the chunks pulled from
    // the io buffer
    exec<ProgType>(instance, batch_count, run_count);
    hipDeviceSynchronize();
    host::check_error();
  } while (!instance.complete());
}